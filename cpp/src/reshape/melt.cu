/* Copyright 2018 NVIDIA Corporation.  All rights reserved. */

#include "dataframe/cudf_table.cuh"
#include <cudf.h>

#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <vector>

void gdf_melt_length(size_t num_id_cols, size_t num_value_cols, size_t old_rows,
    size_t * new_rows, size_t * new_cols)
{
    *new_rows = old_rows * num_value_cols;
    *new_cols = num_id_cols + 2;
}

gdf_error gdf_melt(gdf_column *id_columns[], size_t num_id_cols, gdf_column *value_columns[], size_t num_value_cols,
    gdf_column *out_columns[])
{
    // Calculate dimensions
    gdf_size_type old_length = 0;
    if (num_id_cols != 0)
    {
        old_length = id_columns[0]->size;
    }
    else if (num_value_cols != 0)
    {
        old_length = value_columns[0]->size;
    }
    else
    {
        return GDF_DATASET_EMPTY;
    }
    gdf_size_type new_length = num_value_cols * old_length;

    // Step 1: tile id_columns
    size_t idx; // persist this outside loop to use for indexing out_columns
    for(idx = 0; idx < num_id_cols; idx++)
    {
        // step 1.1: make vector of pointers to same id_col
        auto cols_to_concat = std::vector<gdf_column *>(num_value_cols, id_columns[idx]);
        // step 1.2: Concat these into one output column
        gdf_error result = gdf_column_concat(out_columns[idx], cols_to_concat.data(), cols_to_concat.size());
        if (GDF_SUCCESS != result) return result;
    }

    // Step 2: Add variable column
    // num_value_cols number of categories
    // repeat each category old_length times.
    void * out_var_col_data = out_columns[idx]->data;
    auto make_category_column = [=] __device__ (gdf_size_type i)
    {
        int8_t * data = reinterpret_cast<decltype(data)> (out_var_col_data);
        data[i] = i / old_length;
    };
    thrust::for_each(thrust::counting_iterator<int>(0),
                    thrust::counting_iterator<int>(new_length),
                    make_category_column);
    CUDA_TRY( hipGetLastError() );
    idx++;

    // Step 3: Add values column
    gdf_error result = gdf_column_concat(out_columns[idx], value_columns, num_value_cols);
    if (GDF_SUCCESS != result) return result;

    return GDF_SUCCESS;
}