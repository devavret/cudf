/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/detail/utilities/device_operators.cuh>

namespace cudf {

__constant__ char max_string_sentinel[5]{"\xF7\xBF\xBF\xBF"};

char const* get_max_string_sentinel() {
  const char* psentinel{nullptr};
  hipGetSymbolAddress((void**)&psentinel, max_string_sentinel);
  return psentinel;
}

} // namespace cudf

